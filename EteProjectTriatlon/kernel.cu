#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <chrono>
#include <thread>
#include <iostream>
#include <map>


struct Player
{
	int velocity;
	int location;
	bool willDisplay;
	int playerIdleTime;
};

const int swimPartLength = 5;
const int bicyclePartLength = 5;
const int runPartLength = 5;

const int delayBetweenParts = 10;

__global__ void calculatePlayersLocation(Player* player, bool* status, int* order, int* currentIndex)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	Player* currPlayer = &player[tid];
	bool* currStatus = &status[tid];
	if (currPlayer->location > swimPartLength + bicyclePartLength + runPartLength)
	{
		return;
	}
	if (currPlayer->playerIdleTime > 0)
	{
		currPlayer->playerIdleTime--;
		return;
	}
	int currentVelocity = currPlayer->velocity;

	if (currPlayer->location < swimPartLength + bicyclePartLength && currPlayer->location > swimPartLength)
	{
		currPlayer->playerIdleTime = 10;
		currentVelocity = currPlayer->velocity * 3;
	}
	else if (currPlayer->location > swimPartLength + bicyclePartLength)
	{
		currPlayer->playerIdleTime = 10;
		currentVelocity = std::ceil(float(currPlayer->velocity) / 3.f);
	}

	currPlayer->location += currentVelocity;
	if (currPlayer->location > swimPartLength + bicyclePartLength + runPartLength)
	{
		int index = atomicAdd(currentIndex, 1);
		int* currOrder = &order[index];
		*currOrder = tid;
		*currStatus = true;
	}
	if (currPlayer->willDisplay)
		printf("Player %d location: %d\n", tid, currPlayer->location);
}

void initializePlayers(Player* players, std::uniform_int_distribution<std::mt19937::result_type>& dist, std::mt19937& rng, int playerCount, std::vector<std::map<int, int>> playersToDisplay, std::vector<int> groupsToDisplay)
{
	for (int i = 0; i < playerCount; i++)
	{
		players[i].velocity = dist(rng);
		players[i].location = 0;
		players[i].willDisplay = false;
		players[i].playerIdleTime = 0;

		if (std::any_of(groupsToDisplay.begin(), groupsToDisplay.end(), [i](int group) { return i / 3 == group - 1; }))
		{
			players[i].willDisplay = true;
			players[i + 1].willDisplay = true;
			players[i + 1].velocity = dist(rng);
			players[i + 1].location = 0;
			players[i + 1].playerIdleTime = 0;
			players[i + 2].willDisplay = true;
			players[i + 2].velocity = dist(rng);
			players[i + 2].location = 0;
			players[i + 2].playerIdleTime = 0;
			i = i + 2;
			std::cout << "Group " << i / 3 << " will be displayed" << std::endl;
		}
		else if (std::any_of(playersToDisplay.begin(), playersToDisplay.end(), [i](std::map<int, int>& player) { return player.find(i) != player.end(); }))
		{
			players[i].willDisplay = true;
			for (const auto& player : playersToDisplay) {
				if (player.find(i) != player.end()) {
					int groupNumber = player.at(i);
					std::cout << "Player " << i << " from group " << groupNumber << " will be displayed" << std::endl;
				}
			}
		}
	}
}

int main(int argc, char* argv[])
{
	std::vector<std::map<int, int>> playersToDisplay;
	std::vector<int> groupsToDisplay;

	if (argc > 1) {
		for (int i = 1; i < argc; i++) {
			int tempPlayer = -1;
			int tempGroup = -1;
			if (std::strcmp(argv[i], "-P") == 0) {
				if (i + 1 < argc) {
					std::string playerArg(argv[i + 1]);
					size_t dashPos = playerArg.find('-');
					if (dashPos == std::string::npos || dashPos == 0 || dashPos == playerArg.length() - 1) {
						std::cerr << "Error: Invalid player format. Please use GROUP-INDEX format, e.g., 3-2" << std::endl;
						return 1;
					}
					std::string groupStr = playerArg.substr(0, dashPos);
					std::string indexStr = playerArg.substr(dashPos + 1);
					int group = std::stoi(groupStr);
					int index = std::stoi(indexStr);
					if (group <= 0 || group > 300 || index <= 0 || index > 3) {
						std::cerr << "Error: Invalid player or group number" << std::endl;
						return 1;
					}
					tempPlayer = index;
					tempGroup = group;
					i++;
				}
				else {
					std::cerr << "Error: Missing GROUP-INDEX after -P" << std::endl;
					return 1;
				}
			}
			else if (std::strcmp(argv[i], "-G") == 0) {

				if (i + 1 < argc) {
					if (std::atoi(argv[i + 1]) <= 0 || std::atoi(argv[i + 1]) > 300)
					{
						std::cerr << "Error: Invalid group number" << std::endl;
						return 1;
					}
					tempGroup = std::atoi(argv[i + 1]);
					i++;
				}
				else {
					std::cerr << "Error: Missing group number after -G" << std::endl;
					return 1;
				}
			}
			else {
				std::cerr << "Error: Invalid argument " << argv[i] << std::endl;
				return 1;
			}
			if (tempPlayer != -1 && tempGroup != -1) {
				playersToDisplay.emplace_back(std::map<int, int>{{tempPlayer, tempGroup}});
			}
			else if (tempGroup != -1)
			{
				groupsToDisplay.push_back(tempGroup);
			}
		}
	}
	else {
		std::cerr << "Usage: program.exe -G <group_number> -P <player_number>, -G <group_number> -P <player_number>, ..." << std::endl;
		std::cerr << "Group number should be between 0 and 300, player number should be between 1 and 3" << std::endl;
		return 1;
	}

	const int GROUP_SIZE = 3;
	const int GROUP_COUNT = 300;

	const int NUM_THREAD = GROUP_SIZE;
	const int NUM_BLOCKS = GROUP_COUNT;

	std::random_device dev;
	std::mt19937 rng(dev());
	std::uniform_int_distribution<std::mt19937::result_type> dist5(1, 5);


	size_t bytesPlayer = NUM_THREAD * NUM_BLOCKS * sizeof(Player);
	size_t bytesBool = NUM_THREAD * NUM_BLOCKS * sizeof(bool);
	size_t bytesOrder = NUM_THREAD * NUM_BLOCKS * sizeof(int);
	Player* d_a;
	bool* d_b;
	int* d_order;
	int* d_currentIndex;

	bool* h_b = (bool*)malloc(bytesBool);
	int* h_order = (int*)malloc(bytesOrder);
	memset(h_b, 0, bytesBool);
	memset(h_order, 0, bytesOrder);
	Player* h_a = (Player*)malloc(bytesPlayer);

	initializePlayers(h_a, dist5, rng, GROUP_SIZE * GROUP_COUNT, playersToDisplay, groupsToDisplay);

	hipMalloc(&d_a, bytesPlayer);
	hipMalloc(&d_b, bytesBool);
	hipMalloc(&d_order, bytesOrder);
	hipMalloc(&d_currentIndex, sizeof(int));

	hipMemcpy(d_a, h_a, bytesPlayer, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytesBool, hipMemcpyHostToDevice);
	hipMemcpy(d_order, h_order, bytesOrder, hipMemcpyHostToDevice);
	bool displayedFirstOne = false;
	while (true)
	{
		calculatePlayersLocation << < NUM_BLOCKS, NUM_THREAD >> > (d_a, d_b, d_order, d_currentIndex);
		hipMemcpy(h_b, d_b, bytesBool, hipMemcpyDeviceToHost);
		bool anyFalse = false;
		bool anyTrue = std::any_of(h_b, h_b + GROUP_COUNT * GROUP_SIZE, [](bool val) { return val; });
		if (anyTrue && !displayedFirstOne)
		{
			hipMemcpy(h_a, d_a, bytesPlayer, hipMemcpyDeviceToHost);
			for(int i = 0; i < GROUP_COUNT * GROUP_SIZE; i++)
			{
				printf("Player %d location: %d\n", i, h_a[i].location);
			}
			displayedFirstOne = true;
		}
		bool allTrue = std::all_of(h_b, h_b + GROUP_COUNT * GROUP_SIZE, [](bool val) { return val; });
		if (allTrue)
			break;
		std::cout << "Not all players finished yet" << std::endl;
		std::this_thread::sleep_for(std::chrono::seconds(1));
	}
	hipMemcpy(h_order, d_order, bytesOrder, hipMemcpyDeviceToHost);
	hipMemcpy(h_a, d_a, bytesPlayer, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_order);

	for(int i = 0; i < GROUP_COUNT * GROUP_SIZE; i++)
	{
		printf("Player %d finished at %d\n", h_order[i], i );
	}

	return 0;
}